#include "hip/hip_runtime.h"
#include "kernel.h"
#include <stdio.h>
#define TX 32
#define TY 32
#define LEN 5.f
#define STEPNUMBER 2

// scale coordinates onto [-LEN, LEN]
__device__
float scale(int i, int w) { return 2*LEN*(((1.f*i)/w) - 0.5f); }

// Newton's Method for Finding Roots
__device__
float fx(float x) {
  return x*x*x - x;
}

__device__
float fxprime(float x) {
  return 3*x*x - 1;
}

__device__
float2 newton(float x, float y, int stepNumber) {
  float dx = 0.f;
  for (float step = 0; step < stepNumber; step += 1){
    dx = fxprime(x);
    x += -(y/dx);
    y = fx(x);
  }
  return make_float2(x,y);
}

__device__
unsigned char clip(float x){ return x > 255 ? 255 : (x < 0 ? 0 : x); }

// kernel function to compute decay and shading
__global__
void stabImageKernel(uchar4 *d_out, int w, int h, float p, int s) {
  const int c = blockIdx.x*blockDim.x + threadIdx.x;
  const int r = blockIdx.y*blockDim.y + threadIdx.y;
  if ((c >= w) || (r >= h)) return; // Check if within image bounds
  const int i = c + r*w; // 1D indexing
  const float x0 = scale(c, w);
  const float y0 = scale(r, h);
  const float dist_0 = sqrt(x0*x0);
  const float2 pos = newton(x0, y0, STEPNUMBER);
  const float dist_f = sqrt(pos.x*pos.x);
  // assign colors based on distance from origin
  const float dist_r = dist_f/dist_0;
  d_out[i].x = clip(dist_r*255); // red ~ growth
  d_out[i].y = ((c == w/2) || (r == h/2)) ? 255 : 0; // axes
  d_out[i].z = clip((1/dist_r)*255);  // blue - 1/growth
  d_out[i].w = 255;

}

void kernelLauncher(uchar4 *d_out, int w, int h, float p, int s) {
  const dim3 blockSize(TX, TY);
  const dim3 gridSize = dim3((w + TX - 1)/TX, (h + TY - 1)/TY);
  stabImageKernel<<<gridSize, blockSize>>>(d_out, w, h, p, s);
}
