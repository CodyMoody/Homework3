#include "hip/hip_runtime.h"
#include "kernel.h"
#include <stdio.h>
#define TPB 128

// Component Multiplication
__global__ void componentMultKernel(float *d_out, float *d_a, float *d_b)
{
  const int i = blockIdx.x*blockDim.x + threadIdx.x;
  d_out[i] = d_a[i] * d_b[i];
}

// Bad Sum
__global__ void sumKernelBad(float *accum, float *d_in, int size)
{
  const int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i >= size) return;
  *accum += d_in[i];
}

// Atomic Add Sum
__global__ void sumKernel(float * accum, const float *a, int size)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i >= size) return;
  atomicAdd(accum, a[i]); // *accum += a[i]
}


// Atomic Add Shared Sum -- Inspired by book example
__global__ void sumKernelShared(float *d_res, const float *d_a, const float *d_b, int n)
{
  const int idx = threadIdx.x*blockDim.x + blockIdx.x;
  if (idx >= n) return;
  const int s_idx = threadIdx.x;

  __shared__ float s_prod[TPB];
  s_prod[s_idx] = d_a[idx] * d_b[idx];
  __syncthreads();
  
  if (s_idx == 0) {
    float blockSum = 0;
    for (int j = 0; j < blockDim.x; ++j) {
      blockSum += s_prod[j];
    }
    //printf("Block_%i, blockSum = %f\n", blockIdx.x, blockSum);
    //*d_res += blockSum;
    atomicAdd(d_res,blockSum);
  }
}

// Problem One Full Encompassing Kernel Launcher
void dotProductLauncher(float *resultGPU, float *resultGPUmem, float *resultBAD, float *time1, float *time2, const float *a, const float *b, int size)
{
  float *d_a = 0;
  float *d_b = 0;

  float *d_mult = 0;
  float *d_accum1 = 0;
  float *d_accum2 = 0;
  float *d_accum3 = 0;
  hipMalloc(&d_accum1, sizeof(float));
  hipMalloc(&d_accum2, sizeof(float));
  hipMalloc(&d_accum3, sizeof(float));
  hipMalloc(&d_mult, size*sizeof(float));
  hipMemset(d_accum1, 0, sizeof(float));
  hipMemset(d_accum2, 0, sizeof(float));
  hipMemset(d_accum3, 0, sizeof(float));

  hipMalloc(&d_a, size * sizeof(float));
  hipMalloc(&d_b, size * sizeof(float));
  hipMemcpy(d_a, a, size * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, size * sizeof(float), hipMemcpyHostToDevice);

  // For timing the kernels
  hipEvent_t startKernelMult, stopKernelMult;
  hipEvent_t startKernelGPU, stopKernelGPU;
  hipEvent_t startKernelGPUshared, stopKernelGPUshared;
  hipEventCreate(&startKernelMult);
  hipEventCreate(&stopKernelMult);
  hipEventCreate(&startKernelGPU);
  hipEventCreate(&stopKernelGPU);
  hipEventCreate(&startKernelGPUshared);
  hipEventCreate(&stopKernelGPUshared);

  hipEventRecord(startKernelMult);
  componentMultKernel<<<(size + TPB - 1)/TPB, TPB>>>(d_mult, d_a, d_b);
  hipEventRecord(stopKernelMult);

  // Atomic Add (GOOD) Computation
  hipEventRecord(startKernelGPU);
  sumKernel<<<(size + TPB - 1)/TPB, TPB>>>(d_accum1, d_mult, size);
  hipEventRecord(stopKernelGPU);

  hipMemcpy(resultGPU, d_accum1, sizeof(float), hipMemcpyDeviceToHost);

  // Atomic Add Computation with Shared Memory
  hipEventRecord(startKernelGPUshared);
  sumKernelShared<<<(size + TPB - 1)/TPB, TPB>>>(d_accum2, d_a, d_b, size);
  hipEventRecord(stopKernelGPUshared);

  hipMemcpy(resultGPUmem, d_accum2, sizeof(float), hipMemcpyDeviceToHost);
  
  // BAD Computation
  sumKernelBad<<<(size + TPB - 1)/TPB, TPB>>>(d_accum3, d_mult, size);

  hipMemcpy(resultBAD, d_accum3, sizeof(float), hipMemcpyDeviceToHost);

  // Synchronize Cuda Events
  hipEventSynchronize(stopKernelMult);
  hipEventSynchronize(stopKernelGPU);
  hipEventSynchronize(stopKernelGPUshared);

  float multTimeInMs = 0;
  float gpuTimeInMs = 0;
  float gpuSharedTimeInMs = 0;

  hipEventElapsedTime(&multTimeInMs, startKernelMult, stopKernelMult);
  hipEventElapsedTime(&gpuTimeInMs, startKernelGPU, stopKernelGPU);
  hipEventElapsedTime(&gpuSharedTimeInMs, startKernelGPUshared, stopKernelGPUshared);

  gpuTimeInMs += multTimeInMs;

  printf("GPU Time in ms: %f\n", gpuTimeInMs);
  printf("GPU Shared Memory Time in ms: %f\n", gpuSharedTimeInMs);

  hipFree(d_accum1);
  hipFree(d_accum2);
  hipFree(d_accum3);
  hipFree(d_mult);
  hipFree(d_a);
  hipFree(d_b);
}
